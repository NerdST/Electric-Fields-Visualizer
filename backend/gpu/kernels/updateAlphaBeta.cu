#include <hip/hip_runtime.h>


__global__ void updateAlphaBetaKernel(uint8_t *materialField,
                                      float *alphaBetaField, float dt,
                                      float cellSize, int width, int height,
                                      int depth) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z * blockDim.z + threadIdx.z;

  if (x >= width || y >= height || z >= depth)
    return;

  int idx = (z * height + y) * width + x;
  int matIdx = idx * 4;

  // Material properties (normalized [0,1])
  float permeability = materialField[matIdx] / 255.0f;
  float permittivity = materialField[matIdx + 1] / 255.0f;
  float conductivity = materialField[matIdx + 2] / 255.0f;

  // Calculate alpha and beta for electric field
  float cEl = conductivity * dt / (2.0f * permeability);
  float dEl = 1.0f / (1.0f + cEl);
  float alphaEl = (1.0f - cEl) * dEl;
  float betaEl = dt / (permeability * cellSize) * dEl;

  // Calculate alpha and beta for magnetic field
  float cMag = conductivity * dt / (2.0f * permittivity);
  float dMag = 1.0f / (1.0f + cMag);
  float alphaMag = (1.0f - cMag) * dMag;
  float betaMag = dt / (permittivity * cellSize) * dMag;

  int outIdx = idx * 4;
  alphaBetaField[outIdx] = alphaEl;
  alphaBetaField[outIdx + 1] = betaEl;
  alphaBetaField[outIdx + 2] = alphaMag;
  alphaBetaField[outIdx + 3] = betaMag;
}

extern "C" void launchUpdateAlphaBeta(void *materialField, void *alphaBetaField,
                                      const float *params, int w, int h,
                                      int d) {
  float dt = params[0];
  float cellSize = params[1];

  dim3 blockSize(8, 8, 8);
  dim3 gridSize((w + blockSize.x - 1) / blockSize.x,
                (h + blockSize.y - 1) / blockSize.y,
                (d + blockSize.z - 1) / blockSize.z);

  updateAlphaBetaKernel<<<gridSize, blockSize>>>(
      (uint8_t *)materialField, (float *)alphaBetaField, dt, cellSize, w, h, d);

  hipDeviceSynchronize();
}
